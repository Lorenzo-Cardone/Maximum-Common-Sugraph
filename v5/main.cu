#include "hip/hip_runtime.h"
#include <argp.h>
#include <limits.h>
#include <locale.h>
#include <stdbool.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#include "graph.h"

#define L   0
#define R   1
#define LL  2
#define RL  3
#define ADJ 4
#define P   5
#define W   6
#define IRL 7

#define BDS 8

#define START 0
#define END 1

#define MIN(a, b) (a < b)? a : b

#define N_BLOCKS 64
#define BLOCK_SIZE 512
#define MAX_GRAPH_SIZE 64
#define checkCudaErrors(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

typedef unsigned char uchar;
typedef unsigned int uint;

__constant__ uchar d_adjmat0[MAX_GRAPH_SIZE][MAX_GRAPH_SIZE];
__constant__ uchar d_adjmat1[MAX_GRAPH_SIZE][MAX_GRAPH_SIZE];
__constant__ uchar d_n0;
__constant__ uchar d_n1;

uchar adjmat0[MAX_GRAPH_SIZE][MAX_GRAPH_SIZE];
uchar adjmat1[MAX_GRAPH_SIZE][MAX_GRAPH_SIZE];
uchar n0;
uchar n1;

uint __gpu_level = 5;
struct timespec start;

static struct argp_option options[] = {
		{ "verbose", 'v', 0, 0, "Verbose output" },
		{ "lad", 'l', 0, 0, "Read LAD format"},
		{ "timeout", 't', "timeout", 0, "Set timeout of TIMEOUT milliseconds"},
		{ "connected", 'c', 0, 0, "Solve max common CONNECTED subgraph problem" },
		{ 0 }
};

static char doc[] = "Find a maximum isomorphic graph";
static char args_doc[] = "FILENAME1 FILENAME2";
static struct {
	bool verbose;
	bool lad;
	bool connected;
	int timeout;
	char *filename1;
	char *filename2;
	int arg_num;
} arguments;
void set_default_arguments() {
	arguments.verbose = false;
	arguments.lad = false;
	arguments.timeout = 0;
	arguments.connected = false;
	arguments.filename1 = NULL;
	arguments.filename2 = NULL;
	arguments.arg_num = 0;
}
static error_t parse_opt(int key, char *arg, struct argp_state *state) {
	switch (key) {
	case 'v':
		arguments.verbose = true;
		break;
	case 't':
		arguments.timeout = strtol(arg, NULL, 10);
		break;
	case 'l':
		arguments.lad = true;
		break;
	case 'c':
		arguments.connected = true;
		break;
	case ARGP_KEY_ARG:
		if (arguments.arg_num == 0) {
			arguments.filename1 = arg;
		} else if (arguments.arg_num == 1) {
			arguments.filename2 = arg;
		} else {
			argp_usage(state);
		}
		arguments.arg_num++;
		break;
	case ARGP_KEY_END:
		if (arguments.arg_num == 0)
			argp_usage(state);
		break;
	default:
		return ARGP_ERR_UNKNOWN;
	}
	return 0;
}

__host__ __device__
void uchar_swap(uchar *a, uchar *b){
	uchar tmp = *a;
	*a = *b;
	*b = tmp;
}

__host__ __device__
uchar select_next_v(uchar *left, uchar *bd){
	uchar min = UCHAR_MAX, idx = UCHAR_MAX;
	if(bd[RL] != bd[IRL])
		return left[bd[L] + bd[LL]];
	for (uchar i = 0; i < bd[LL]; i++)
		if (left[bd[L] + i] < min) {
			min = left[bd[L] + i];
			idx = i;
		}
	uchar_swap(&left[bd[L] + idx], &left[bd[L] + bd[LL] - 1]);
	bd[LL]--;
	bd[RL]--;
	return min;
}


__host__ __device__
uchar select_next_w(uchar *right, uchar *bd) {
	uchar min = UCHAR_MAX, idx = UCHAR_MAX;
	for (uchar i = 0; i < bd[RL]+1; i++)
		if ((right[bd[R] + i] > bd[W] || bd[W] == UCHAR_MAX)
				&& right[bd[R] + i] < min) {
			min = right[bd[R] + i];
			idx = i;
		}
	if(idx == UCHAR_MAX)
		bd[RL]++;
	return idx;
}


__host__  __device__ uchar index_of_next_smallest(const uchar *arr,
		uchar start_idx, uchar len, uchar w) {
	uchar idx = UCHAR_MAX;
	uchar smallest = UCHAR_MAX;
	for (uchar i = 0; i < len; i++) {
		if ((arr[start_idx + i] > w || w == UCHAR_MAX)
				&& arr[start_idx + i] < smallest) {
			smallest = arr[start_idx + i];
			idx = i;
		}
	}
	return idx;
}

__host__  __device__ uchar find_min_value(const uchar *arr, uchar start_idx,
		uchar len) {
	uchar min_v = UCHAR_MAX;
	for (int i = 0; i < len; i++) {
		if (arr[start_idx + i] < min_v)
			min_v = arr[start_idx + i];
	}
	return min_v;
}

__host__ __device__
void remove_from_domain(uchar *arr, const uchar *start_idx, uchar *len,
		uchar v) {
	int i = 0;
	for (i = 0; arr[*start_idx + i] != v; i++)
		;
	uchar_swap(&arr[*start_idx + i], &arr[*start_idx + *len - 1]);
	(*len)--;
}

__host__ __device__
void update_incumbent(uchar cur[][2], uchar inc[][2], uchar cur_pos,
		uchar *inc_pos) {
	if (cur_pos > *inc_pos) {
		*inc_pos = cur_pos;
		for (int i = 0; i < cur_pos; i++) {
			inc[i][L] = cur[i][L];
			inc[i][R] = cur[i][R];
		}
	}
}

// BIDOMAINS FUNCTIONS /////////////////////////////////////////////////////////////////////////////////////////////////
__host__ __device__
void add_bidomain(uchar domains[][BDS], uint *bd_pos, uchar left_i,
		uchar right_i, uchar left_len, uchar right_len, uchar is_adjacent,
		uchar cur_pos) {
	domains[*bd_pos][L] 	= left_i;
	domains[*bd_pos][R] 	= right_i;
	domains[*bd_pos][LL] 	= left_len;
	domains[*bd_pos][RL] 	= right_len;
	domains[*bd_pos][ADJ] 	= is_adjacent;
	domains[*bd_pos][P] 	= cur_pos;
	domains[*bd_pos][W] 	= UCHAR_MAX;
	domains[*bd_pos][IRL] 	= right_len;

	(*bd_pos)++;
}

__host__  __device__ uint calc_bound(uchar domains[][BDS], uint bd_pos,
		uint cur_pos, uint *bd_n) {
	uint bound = 0;
	int i;
	for (i = bd_pos - 1; i >= 0 && domains[i][P] == cur_pos; i--)
		bound += MIN(domains[i][LL], domains[i][IRL]);
	*bd_n = bd_pos - 1 - i;
	return bound;
}

__host__  __device__ uchar partition(uchar *arr, uchar start, uchar len,
		const uchar *adjrow) {
	uchar i = 0;
	for (uchar j = 0; j < len; j++) {
		if (adjrow[arr[start + j]]) {
			uchar_swap(&arr[start + i], &arr[start + j]);
			i++;
		}
	}
	return i;
}

__host__  __device__
uchar find_min_value(uchar *arr, uchar start_idx, uchar len){
	uchar min_v = UCHAR_MAX;
	for(int i = 0; i < len; i++){
		if(arr[start_idx+i] < min_v)
			min_v = arr[start_idx + i];
	}
	return min_v;
}

__host__  __device__
void select_bidomain(uchar domains[][BDS], uint bd_pos,  uchar *left, int current_matching_size, bool connected){
	int i;
	uint min_size = UINT_MAX;
	uint min_tie_breaker = UINT_MAX;
	uint best = UINT_MAX;
	uchar *bd;
	for (i = bd_pos - 1, bd = &domains[i][L]; i >= 0 && bd[P] == current_matching_size; i--, bd = &domains[i][L]) {
		if (connected && current_matching_size>0 && !bd[ADJ]) continue;
		int len = bd[LL] > bd[RL] ? bd[LL] : bd[RL];
		if (len < min_size) {
			min_size = len;
			min_tie_breaker = find_min_value(left, bd[L], bd[LL]);
			best = i;
		} else if (len == min_size) {
			int tie_breaker = find_min_value(left, bd[L], bd[LL]);
			if (tie_breaker < min_tie_breaker) {
				min_tie_breaker = tie_breaker;
				best = i;
			}
		}
	}
	if(best != UINT_MAX && best != bd_pos-1){
		uchar tmp[BDS];
		for(i = 0; i < BDS; i++) tmp[i] = domains[best][i];
		for(i = 0; i < BDS; i++) domains[best][i] = domains[bd_pos-1][i];
		for(i = 0; i < BDS; i++) domains[bd_pos-1][i] = tmp[i];

	}
}



__device__
void d_generate_next_domains(uchar domains[][BDS], uint *bd_pos, uint cur_pos, uchar *left, uchar *right, uchar v, uchar w, uint inc_pos) {
	int i;
	uint bd_backup = *bd_pos;
	uint bound = 0;
	uchar *bd;
	for (i = *bd_pos - 1, bd = &domains[i][L]; i >= 0 && bd[P] == cur_pos - 1; i--, bd = &domains[i][L]) {

		uchar l_len = partition(left, bd[L], bd[LL], d_adjmat0[v]);
		uchar r_len = partition(right, bd[R], bd[RL], d_adjmat1[w]);

		if (bd[LL] - l_len && bd[RL] - r_len) {
			add_bidomain(domains, bd_pos, bd[L] + l_len, bd[R] + r_len, bd[LL] - l_len, bd[RL] - r_len, bd[ADJ], (uchar) (cur_pos));
			bound += MIN(bd[LL] - l_len, bd[RL] - r_len);
		}
		if (l_len && r_len) {
			add_bidomain(domains, bd_pos, bd[L], bd[R], l_len, r_len, true, (uchar) (cur_pos));
			bound += MIN(l_len, r_len);
		}
	}
	if (cur_pos + bound <= inc_pos)
		*bd_pos = bd_backup;
}

__global__
void d_mcs(uchar *args, uint n_threads, uchar a_size, uint *args_i, uint actual_inc, uchar *device_solutions, uint max_sol_size, uint last_arg, bool verbose, bool connected) {
	uint my_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	uchar cur[MAX_GRAPH_SIZE][2], incumbent[MAX_GRAPH_SIZE][2],
	domains[MAX_GRAPH_SIZE * 5][BDS], left[MAX_GRAPH_SIZE],
	right[MAX_GRAPH_SIZE], v, w;
	uint bd_pos = 0, bd_n = 0;
	uchar inc_pos = 0;
	__shared__ uint sh_inc;
	sh_inc = actual_inc;
	__syncthreads();
	if (my_idx < n_threads) {
		for (int i = args_i[my_idx]; i < last_arg && ( my_idx < n_threads-1 &&  i < args_i[my_idx +1]);) {
			add_bidomain(domains, &bd_pos, args[i++], args[i++], args[i++], args[i++], args[i++], args[i++]);
			for (int p = 0; p < domains[bd_pos - 1][P]; p++)
				cur[p][L] = args[i++];
			for (int p = 0; p < domains[bd_pos - 1][P]; p++)
				cur[p][R] = args[i++];
			for (int l = 0; l < d_n0; l++)
				left[l] = args[i++];
			for (int r = 0; r < d_n1; r++)
				right[r] = args[i++];
		}
		while (bd_pos > 0) {
			uchar *bd = &domains[bd_pos - 1][L];
			if (calc_bound(domains, bd_pos, bd[P], &bd_n) + bd[P] <= sh_inc	|| (bd[LL] == 0 && bd[RL] == bd[IRL])) {
				bd_pos--;
			} else {
				select_bidomain(domains, bd_pos, left, domains[bd_pos - 1][P], connected);
				if (bd[RL] == bd[IRL]) {
					v = find_min_value(left, bd[L], bd[LL]);
					remove_from_domain(left, &bd[L], &bd[LL], v);
					bd[RL]--;
				} else v = left[bd[L] + bd[LL]];
				if ((bd[W] = index_of_next_smallest(right, bd[R], bd[RL] + (uchar) 1, bd[W])) == UCHAR_MAX) {
					bd[RL]++;
				} else {
					w = right[bd[R] + bd[W]];
					right[bd[R] + bd[W]] = right[bd[R] + bd[RL]];
					right[bd[R] + bd[RL]] = w;
					bd[W] = w;
					cur[bd[P]][L] = v;
					cur[bd[P]][R] = w;
					update_incumbent(cur, incumbent, bd[P] + 1, &inc_pos);
					atomicMax(&sh_inc, inc_pos);
					d_generate_next_domains(domains, &bd_pos, bd[P] + 1, left, right, v, w, inc_pos);
				}
			}
		}
	}
	device_solutions[blockIdx.x* max_sol_size] = 0;

	__syncthreads();
	if (atomicCAS(&sh_inc, inc_pos, 0) == inc_pos && inc_pos > 0) {
		if(verbose) printf("Th_%d found new solution of size %d\n", my_idx, inc_pos);
		bd_pos = 0;
		device_solutions[blockIdx.x* max_sol_size + bd_pos++] = inc_pos;
		for (int i = 0; i < inc_pos; i++)
			device_solutions[blockIdx.x* max_sol_size + bd_pos++] = incumbent[i][L];
		for (int i = 0; i < inc_pos; i++)
			device_solutions[blockIdx.x* max_sol_size + bd_pos++] = incumbent[i][R];
	}
}

double compute_elapsed_sec(struct timespec strt){
	struct timespec now;
	double time_elapsed;

	clock_gettime(CLOCK_MONOTONIC, &now);
	time_elapsed = (now.tv_sec - strt.tv_sec);
	time_elapsed += (double)(now.tv_nsec - strt.tv_nsec) / 1000000000.0;

	return time_elapsed;
}

static void CheckCudaErrorAux(const char *file, unsigned line,
		const char *statement, hipError_t err) {
	if (err == hipSuccess)
		return;
	fprintf(stderr, "%s returned %s(%d) at %s:%d\n", statement,
			hipGetErrorString(err), err, file, line);
	exit(1);
}

void move_graphs_to_gpu(graph_t *g0, graph_t *g1) {
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_n0), &g0->n, sizeof(uchar)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_n1), &g1->n, sizeof(uchar)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_adjmat0), adjmat0, MAX_GRAPH_SIZE*MAX_GRAPH_SIZE));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_adjmat1), adjmat1, MAX_GRAPH_SIZE*MAX_GRAPH_SIZE));
}


void h_generate_next_domains(uchar domains[][BDS], uint *bd_pos, uint cur_pos,
		uchar *left, uchar *right, uchar v, uchar w, uint inc_pos) {
	int i;
	uint bd_backup = *bd_pos;
	uint bound = 0;
	uchar *bd;
	for (i = *bd_pos - 1, bd = &domains[i][L]; i >= 0 && bd[P] == cur_pos - 1;
			i--, bd = &domains[i][L]) {

		uchar l_len = partition(left, bd[L], bd[LL], adjmat0[v]);
		uchar r_len = partition(right, bd[R], bd[RL], adjmat1[w]);

		if (bd[LL] - l_len && bd[RL] - r_len) {
			add_bidomain(domains, bd_pos, bd[L] + l_len, bd[R] + r_len,
					bd[LL] - l_len, bd[RL] - r_len, bd[ADJ], (uchar) (cur_pos));
			bound += MIN(bd[LL] - l_len, bd[RL] - r_len);
		}
		if (l_len && r_len) {
			add_bidomain(domains, bd_pos, bd[L], bd[R], l_len, r_len, true,
					(uchar) (cur_pos));
			bound += MIN(l_len, r_len);
		}
	}
	if (cur_pos + bound <= inc_pos)
		*bd_pos = bd_backup;
}


bool check_sol(graph_t *g0, graph_t *g1, uchar sol[][2], uint sol_len) {
	bool *used_left = (bool*) calloc(g0->n, sizeof *used_left);
	bool *used_right = (bool*) calloc(g1->n, sizeof *used_right);
	for (int i = 0; i < sol_len; i++) {
		if (used_left[sol[i][L]]) {
			printf("node %d of g0 used twice\n", used_left[sol[i][L]]);
			return false;
		}
		if (used_right[sol[i][R]]) {
			printf("node %d of g1 used twice\n", used_right[sol[i][L]]);
			return false;
		}
		used_left[sol[i][L]] = true;
		used_right[sol[i][R]] = true;
		if (g0->label[sol[i][L]] != g1->label[sol[i][R]]) {
			printf("g0:%d and g1:%d have different labels\n", sol[i][L],
					sol[i][R]);
			return false;
		}
		for (int j = i + 1; j < sol_len; j++) {
			if (g0->adjmat[sol[i][L]][sol[j][L]]
			                          != g1->adjmat[sol[i][R]][sol[j][R]]) {
				printf("g0(%d-%d) is different than g1(%d-%d)\n", sol[i][L],
						sol[j][L], sol[i][R], sol[j][R]);
				return false;
			}
		}
	}
	return true;
}

static struct argp argp = { options, parse_opt, args_doc, doc };

void launch_kernel(uchar *args, uint n_threads, uchar a_size, uint sol_size, uint *args_i,
		uchar incumbent[][2], uchar *inc_pos, uint total_args_size, uint last_arg) {
	uchar *device_args;
	uchar *device_solutions;
	uchar *host_solutions;
	uint *device_args_i;
	uint max_sol_size = 1 + 2 * (MIN(n0, n1));
	struct timespec sleep;
	sleep.tv_sec = 0;
	sleep.tv_nsec = 2000;
	hipEvent_t stop;

	host_solutions = (uchar*) malloc(N_BLOCKS * max_sol_size * sizeof *host_solutions);

	checkCudaErrors(hipEventCreate(&stop));

	checkCudaErrors(hipMalloc(&device_args, total_args_size * sizeof *device_args));
	checkCudaErrors(hipMalloc(&device_solutions, N_BLOCKS * max_sol_size * sizeof *device_solutions));


	checkCudaErrors(hipMemcpy(device_args, args, total_args_size * sizeof *device_args, hipMemcpyHostToDevice));

	checkCudaErrors(hipMalloc(&device_args_i, N_BLOCKS * BLOCK_SIZE * sizeof *device_args_i));
	checkCudaErrors(hipMemcpy(device_args_i, args_i, N_BLOCKS * BLOCK_SIZE * sizeof *device_args_i, hipMemcpyHostToDevice));

	if(arguments.verbose) printf("Launching kernel...\n");

	d_mcs<<<N_BLOCKS, BLOCK_SIZE>>>(device_args, n_threads, a_size, device_args_i, *inc_pos, device_solutions, max_sol_size, last_arg, arguments.verbose, arguments.connected);
	checkCudaErrors(hipEventRecord(stop));

	while(hipEventQuery(stop) == hipErrorNotReady){
		nanosleep(&sleep, NULL);
		if(arguments.timeout && compute_elapsed_sec(start) > arguments.timeout)
			return;
	}

	if(arguments.verbose) printf("Kernel executed...\n");

	checkCudaErrors(hipMemcpy(host_solutions, device_solutions, N_BLOCKS * max_sol_size * sizeof *device_solutions, hipMemcpyDeviceToHost));

	checkCudaErrors(hipFree(device_args));
	checkCudaErrors(hipFree(device_args_i));

	for(int b = 0; b < N_BLOCKS; b++){
		if (*inc_pos < host_solutions[b*max_sol_size]) {
			*inc_pos = host_solutions[b*max_sol_size];
			for (int i = 1; i < *inc_pos + 1; i++) {
				incumbent[i - 1][L] = host_solutions[b*max_sol_size + i];
				incumbent[i - 1][R] = host_solutions[b*max_sol_size + *inc_pos + i];
				if(arguments.verbose) printf("|%d %d| ", incumbent[i-1][L], incumbent[i-1][R]);
			}if(arguments.verbose) printf("\n");
		}
	}
	free(host_solutions);
}

void *safe_realloc(void* old, uint new_size){
	void *tmp = realloc(old, new_size);
	if (tmp != NULL) return tmp;
	else exit(-1);
}

void mcs(uchar incumbent[][2], uchar *inc_pos) {
	uint bd_pos = 0, bd_n = 0;
	uchar cur[MAX_GRAPH_SIZE][2], domains[MAX_GRAPH_SIZE * 5][BDS], left[n0],
	right[n1], v, w;
	for (uchar i = 0; i < n0; i++)
		left[i] = i;
	for (uchar i = 0; i < n1; i++)
		right[i] = i;
	add_bidomain(domains, &bd_pos, 0, 0, n0, n1, 0, 0);
	//supposing an initial average of 2 domains for thread, it will be reallocated if necessary
	uint args_num = N_BLOCKS * BLOCK_SIZE * 2;
	uint a_size = (BDS - 2 + 2 * __gpu_level + n0 + n1);
	uint sol_size = 1 + 2*(MIN(n0, n1));
	uint args_size = args_num * a_size;

	uint args_i[N_BLOCKS * BLOCK_SIZE];
	uchar *args = (uchar*) malloc(args_size * sizeof *args);
	uint n_args = 0, n_threads = 0;

	while (bd_pos > 0) {
		if (arguments.timeout && compute_elapsed_sec(start) > arguments.timeout) {
			arguments.timeout = -1;
			return;
		}
		uchar *bd = &domains[bd_pos - 1][L];

		if (calc_bound(domains, bd_pos, bd[P], &bd_n) + bd[P] <= *inc_pos || (bd[LL] == 0 && bd[RL] == bd[IRL])) {
			bd_pos--;
			continue;
		}

		if (bd[P] == __gpu_level) {
			if (n_args + bd_n > args_num) {
				args_num  = n_args + bd_n;
				args_size = args_num * a_size;
				args = (uchar*) safe_realloc(args, args_size * sizeof *args);
			}

			args_i[n_threads] = n_args * a_size;

			for (uint b = 0; b < bd_n; b++, n_args++, bd_pos--) {
				uint arg_i = n_args * a_size, i = 0;
				for (i = 0; i < BDS - 2; i++, arg_i++)
					args[arg_i] = domains[bd_pos - 1][i];
				for (i = 0; i < __gpu_level; i++, arg_i++)
					args[arg_i] = cur[i][L];
				for (i = 0; i < __gpu_level; i++, arg_i++)
					args[arg_i] = cur[i][R];
				for (i = 0; i < n0; i++, arg_i++)
					args[arg_i] = left[i];
				for (i = 0; i < n0; i++, arg_i++)
					args[arg_i] = right[i];
			}
			n_threads++;
			if (n_threads == N_BLOCKS * BLOCK_SIZE) {
				launch_kernel(args, n_threads, a_size, sol_size, args_i, incumbent, inc_pos, args_size, n_args*a_size);
				n_threads = 0;
				n_args = 0;
			}
			continue;
		}

		select_bidomain(domains, bd_pos, left, domains[bd_pos - 1][P], arguments.connected);
		if (bd[RL] == bd[IRL]) {
			v = find_min_value(left, bd[L], bd[LL]);
			remove_from_domain(left, &bd[L], &bd[LL], v);
			bd[RL]--;
		} else v = left[bd[L] + bd[LL]];


		if ((bd[W] = index_of_next_smallest(right, bd[R], bd[RL] + (uchar) 1, bd[W])) == UCHAR_MAX) {
			bd[RL]++;
		} else {
			w = right[bd[R] + bd[W]];
			right[bd[R] + bd[W]] = right[bd[R] + bd[RL]];
			right[bd[R] + bd[RL]] = w;

			bd[W] = w;

			cur[bd[P]][L] = v;
			cur[bd[P]][R] = w;

			update_incumbent(cur, incumbent, bd[P] + 1, inc_pos);
			h_generate_next_domains(domains, &bd_pos, bd[P] + 1, left, right, v,
					w, *inc_pos);
		}

	}
	if (n_threads > 0)
		launch_kernel(args, n_threads, a_size, sol_size, args_i, incumbent, inc_pos, args_size, n_args*a_size);
}

int main(int argc, char** argv) {
	set_default_arguments();
	argp_parse(&argp, argc, argv, 0, 0, 0);
	struct timespec finish;
	double time_elapsed;
	char format = arguments.lad ? 'L' : 'B';
	graph_t *g0 = (graph_t*) calloc(1, sizeof *g0);
	readGraph(arguments.filename1, g0, format);
	graph_t *g1 = (graph_t*) calloc(1, sizeof *g1);
	readGraph(arguments.filename2, g1, format);
	g0 = sort_vertices_by_degree(g0,
			(graph_edge_count(g1) > g1->n * (g1->n - 1) / 2));
	g1 = sort_vertices_by_degree(g1,
			(graph_edge_count(g0) > g0->n * (g0->n - 1) / 2));
	int min_size = MIN(g0->n, g1->n);
	n0 = g0->n;
	n1 = g1->n;

	for (int i = 0; i < n0; i++)
		for (int j = 0; j < n0; j++)
			adjmat0[i][j] = g0->adjmat[i][j];

	for (int i = 0; i < n1; i++)
		for (int j = 0; j < n1; j++)
			adjmat1[i][j] = g1->adjmat[i][j];
	checkCudaErrors(hipDeviceReset());
	move_graphs_to_gpu(g0, g1);
	uchar solution[min_size][2];
	uchar sol_len = 0;
	clock_gettime(CLOCK_MONOTONIC, &start);
	mcs(solution, &sol_len);
	clock_gettime(CLOCK_MONOTONIC, &finish);

	if(arguments.timeout == -1){
		printf("TIMEOUT\n");
	}

	printf("------------------------------------------------------------\n");
	printf("SOLUTION size:%d\nsol: ", sol_len);
	//for (int i = 0; i < g0->n; i++)
	for (int j = 0; j < sol_len; j++)
		//if (solution[j][L] == i)
		printf("|%2d %2d| ", solution[j][L], solution[j][R]);
	printf("\n");

	if (!check_sol(g0, g1, solution, sol_len)) {
		printf("*** Error: Invalid solution\n");
	}
	time_elapsed = (finish.tv_sec - start.tv_sec); // calculating elapsed seconds
	time_elapsed += (double) (finish.tv_nsec - start.tv_nsec) / 1000000000.0; // adding elapsed nanoseconds
	printf(">>> %d - %015.10f\n", sol_len, time_elapsed);

	free_graph(g0);
	free_graph(g1);
	return 0;
}
