#include "hip/hip_runtime.h"
/*
 *  graphISO: Tools to compute the Maximum Common Subgraph between two graphs
 *  Copyright (c) 2019 Stefano Quer
 *  
 *  This program is free software : you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 3 of the License, or
 *  (at your option) any later version.
 *
 *  This program is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with this program.If not, see < http: *www.gnu.org/licenses/>
 */

#include "graph.h"




static void fail(const char* msg) {
    printf("%s\n", msg);
    exit(1);
}

unsigned int* calculate_degrees(graph_t *g) {
    short int size = g->n;
    uint *degree = (uint*)calloc(size, sizeof *degree);
    for (int v = 0; v < g->n; v++)
        for (int w = 0; w < g->n; w++)
            if (g->adjmat[v][w]) degree[v]++;
    return degree;
}

void add_edge(graph_t *g, int v, int w) {
    if (v != w) {
        g->adjmat[v][w] = 1;
        g->adjmat[w][v] = 1;
    } else {
        // To indicate that a vertex has a loop, we set its label to 1
        g->label[v] = 1;
    }
}


unsigned int read_word(FILE *fp) {
	unsigned char a[2];
	if (fread(a, 1, 2, fp) != 2)
		fail("Error reading file.\n");
	return (unsigned int)a[0] | (((unsigned int)a[1]) << 8);
}

// Precondition: *g is already zeroed out
// returns max edge label
void readBinaryGraph(char* filename, graph_t* g) {
    FILE* f;
	int i;
    if ((f=fopen(filename, "rb"))==NULL)
        fail("Cannot open file");

    unsigned int nvertices = read_word(f);
    g->n = nvertices;
    g->label = (uint*)calloc(g->n, sizeof *g->label);
    g->adjmat = (uchar**)calloc(g->n, sizeof *g->adjmat);
    for(i = 0; i < g->n; i++)
    	g->adjmat[i] = (uchar*)calloc(g->n, sizeof *g->adjmat[i]);
    printf("%d vertices\n", nvertices);
    
    printf("paolo2");
    for (int i=0; i<nvertices; i++) {
        read_word(f);   // ignore label
    }

    for (int i=0; i<nvertices; i++) {
        int len = read_word(f);
        for (int j=0; j<len; j++) {
            int target = read_word(f);
            read_word(f); // ignore label
            add_edge(g, i, target);
        }
    }
	g->degree = calculate_degrees(g);
    fclose(f);
}

// Precondition: *g is already zeroed out
void readLadGraph(char* filename, graph_t* g) {
    FILE* f;
    int i;
    if ((f=fopen(filename, "r"))==NULL){
        free(g);
        fail((char*)"Cannot open file");
    }
    int nvertices = 0;
    uchar w;
    if (fscanf(f, "%d", &nvertices) != 1)
        fail((char*)"Number of vertices not read correctly.\n");
    g->n = (uchar)nvertices;
    g->label = (uint*)calloc(g->n, sizeof *g->label);
    g->adjmat = (uchar**)calloc(g->n, sizeof *g->adjmat);
    for(i = 0; i < g->n; i++)
        g->adjmat[i] = (uchar*)calloc(g->n, sizeof *g->adjmat[i]);
    for (int i=0; i<nvertices; i++) {
        int edge_count;
        if (fscanf(f, "%d", &edge_count) != 1)
            fail((char*)"Number of edges not read correctly.\n");
        for (int j=0; j<edge_count; j++) {
            if (fscanf(f, "%hhu", &w) != 1)
                fail((char*)"An edge was not read correctly.\n");
            add_edge(g, i, w);
        }
    }
    g->degree = calculate_degrees(g);
    fclose(f);
}

void readGraph(char* filename, graph_t* g, char format) {
	if (format=='L') readLadGraph(filename, g);
	else if (format=='B') readBinaryGraph(filename, g);
	else fail("Unknown graph format\n");
}


graph_t *induced_subgraph(graph_t *g, int *vv) {
    graph_t * subg = (graph_t*)calloc(1, sizeof *subg);
    subg->n = g->n;
    subg->label = (uint*)calloc(g->n, sizeof *subg->label);
    subg->adjmat = (uchar**)calloc(g->n, sizeof *subg->adjmat);
    for (int n = 0; n < g->n; n++) subg->adjmat[n] = (uchar*)calloc(g->n, sizeof *subg->adjmat[n]);
    for (int i = 0; i < subg->n; i++)
        for (int j=0; j < subg->n; j++)
            subg->adjmat[i][j] = g->adjmat[vv[i]][vv[j]];
    for (int i=0; i<subg->n; i++)
        subg->label[i] = g->label[vv[i]];
    subg->degree = calculate_degrees(subg);
    return subg;
}

int graph_edge_count(graph_t *g) {
    int count = 0;
    for (int i=0; i<g->n; i++)
        count += g->degree[i];
    return count;
}

graph_t *sort_vertices_by_degree(graph_t *g, bool ascending ){
    int *vv = (int*)malloc(g->n * sizeof *vv );
    for (int i=0; i<g->n; i++) vv[i] = i;
    if (ascending) {
        INSERTION_SORT(int, vv, g->n, (g->degree[vv[j-1]] > g->degree[vv[j]]))
    } else {
        INSERTION_SORT(int, vv, g->n, (g->degree[vv[j-1]] < g->degree[vv[j]]))
    }
    graph_t *g_sorted = induced_subgraph(g, vv);
    free(vv);
    free_graph(g);
    return g_sorted;
}

void free_graph(graph_t *g){
    for(int i = 0; i < g->n; i++)
        free(g->adjmat[i]);
    free(g->adjmat);
    free(g->label);
    free(g->degree);
    free(g);
    return;
}
